#include "hip/hip_runtime.h"
#include "../../include/cuda/parallel_minmax.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

const int NUMBER_OF_BLOCKS = 1;
const int THREADS_PER_BLOCK = 1;

__global__ void cuda_recursive(int depth)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id == 0)
    {
        printf("Print with depth: %i\n", depth);
    }
    if(depth == 0)
    {
        return;
    }

    cuda_recursive<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(depth - 1);
    hipDeviceSynchronize();
    __syncthreads();
}

__global__ void test()
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id == 0)
    {
        printf("Inside kernel has!");
    }
}

void recursive(int depth)
{
    cuda_recursive<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(depth);
    hipDeviceSynchronize();
}